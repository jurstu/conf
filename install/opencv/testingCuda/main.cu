#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

// CUDA kernel
__global__ void helloWorld() {
    printf("Hello, World from GPU!\\n");
}

int main() {
    // Check for CUDA-capable devices
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA-capable device found!" << std::endl;
        return -1;
    } else {
        std::cout << "CUDA-capable devices found: " << deviceCount << std::endl;
    }

    // Launch the kernel
    helloWorld<<<5, 1>>>();

    // Check for kernel launch errors
    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(kernelErr) << std::endl;
        return -1;
    }

    // Synchronize the device
    hipError_t syncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) {
        std::cerr << "CUDA synchronization error: " << hipGetErrorString(syncErr) << std::endl;
        return -1;
    }

    std::cout << "Hello, World from CPU!" << std::endl;

    return 0;
}

